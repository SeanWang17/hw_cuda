
#include <hip/hip_runtime.h>

__global__
void spotrf_batched_kernel(int n, int batch, float *dA);

////////////////////////////////////////////////////////////////////////////////
extern "C"
void spotrf_batched(int n, int batch, float *dA, hipStream_t stream)
{
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1, 1, 1);
    spotrf_batched_kernel<<<dimGrid, dimBlock, 0, stream>>>(n, batch, dA);
}

////////////////////////////////////////////////////////////////////////////////
__global__
void spotrf_batched_kernel(int N, int batch, float *dA)
{
    int m;
    int n;
    int k;
    int i;

    // Batched Cholesky factorization.
    for (i = 0; i < batch; i++) {

        float *pA = &dA[i*N*N];

        // Single Cholesky factorization.
        for (k = 0; k < N; k++) {

            // Panel factorization.
            pA[k*N+k] = sqrtf(pA[k*N+k]);
            for (m = k+1; m < N; m++)
                pA[k*N+m] /= pA[k*N+k];

            // Update of the trailing submatrix.
            for (n = k+1; n < N; n++)
                for (m = n; m < N; m++)
                    pA[n*N+m] -= (pA[k*N+n]*pA[k*N+m]);
        }
    }
}
